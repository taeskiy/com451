/*******************************************************************************
*
*   probe a comuter for basic info about processing cores and GPU
*
*   compile with:
*
*			nvcc probe2.cu -L/usr/local/cuda/lib64 -I/usr/local/cuda-10.2/targets/x86_64-linux/include -lcuda -lcudart
*
*			(in .tcshrc, please have:)
*			set path = ($path /usr/local/cuda-10.1/bin ./)
*			setenv LD_LIBRARY_PATH /usr/local/lib:/usr/local/cuda-10.1/lib64
*
*******************************************************************************/
#include <stdio.h>


/******************************************************************************/
int main(int argc, char *argv[]){

  hipError_t err;
  err = hipDeviceReset();

  hipDeviceProp_t prop;
  int count;
  err = hipGetDeviceCount(&count);
  if(err != hipSuccess){
    printf("problem getting device count = %s\n", hipGetErrorString(err));
    return 1;
    }
  printf("number of GPU devices: %d\n\n", count);

  for (int i = 0; i< count; i++){
    printf("************ GPU Device: %d ************\n\n", i);
    err = hipGetDeviceProperties(&prop, i);
    if(err != hipSuccess){
      printf("problem getting device properties = %s\n", hipGetErrorString(err));
      return 1;
      }

    printf("\tName: %s\n", prop.name);
    printf( "\tCompute capability: %d.%d\n", prop.major, prop.minor);
    printf( "\tClock rate: %d\n", prop.clockRate );
    printf( "\tDevice copy overlap: " );
      if (prop.deviceOverlap)
        printf( "Enabled\n" );
      else
        printf( "Disabled\n" );
    printf( "\tKernel execition timeout: " );
      if (prop.kernelExecTimeoutEnabled)
        printf( "Enabled\n" );
      else
        printf( "Disabled\n" );
    printf( "--- Memory Information for device %d ---\n", i );
    printf("\tTotal global mem: %ld\n", prop.totalGlobalMem );
    printf("\tTotal constant Mem: %ld\n", prop.totalConstMem );
    printf("\tMax mem pitch: %ld\n", prop.memPitch );
    printf( "\tTexture Alignment: %ld\n", prop.textureAlignment );
    printf("\n");
    printf( "\tMultiprocessor count: %d\n", prop.multiProcessorCount );
    printf( "\tShared mem per processor: %ld\n", prop.sharedMemPerBlock );
    printf( "\tRegisters per processor: %d\n", prop.regsPerBlock );
    printf( "\tThreads in warp: %d\n", prop.warpSize );
    printf( "\tMax threads per block: %d\n", prop.maxThreadsPerBlock );
    printf( "\tMax block dimensions: (%d, %d, %d)\n",
                  prop.maxThreadsDim[0],
                  prop.maxThreadsDim[1],
                  prop.maxThreadsDim[2]);
    printf( "\tMax grid dimensions: (%d, %d, %d)\n",
                  prop.maxGridSize[0],
                  prop.maxGridSize[1],
                  prop.maxGridSize[2]);
    printf("\n");
  }

return 0;
}

/******************************************************************************/
